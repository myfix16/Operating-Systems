﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

__device__ __managed__ u32 gtime = 0;


__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

}



__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	/* Implement write operation here */
}
__device__ void fs_gsys(FileSystem *fs, int op)
{
	/* Implement LS_D and LS_S operation here */
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	/* Implement rm operation here */
}
